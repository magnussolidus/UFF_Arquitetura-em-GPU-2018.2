#include "hip/hip_runtime.h"
#include "trabalho.h"

User *generateUser(char *buffer, int nRatings)
{
  int pos = 0, ratePos = 0, state = 0, size;
  float rateAux = -1.0f;
  User *newUser = (User*)malloc(sizeof(User));
  
  newUser->name = "None";
  newUser->rating = (float*) calloc(nRatings, sizeof(float));
  
  char *tempBuffer = "";

  size = strlen(buffer);

  do {

    if(state == 0 && buffer[pos] == ';')
    {
      tempBuffer = buffer;
      strtok(tempBuffer, ";");
      newUser->name = (char*)malloc(strlen(tempBuffer) * sizeof(char));
      newUser->name = strcpy(newUser->name, tempBuffer);
      state = 1;
    }
    else if(state == 1 && buffer[pos] != ';')
    {
		// ratings
        switch(buffer[pos])
        {
			case '?': 
				newUser->rating[ratePos] = -1;
				break;
				
			default:
				sscanf(&buffer[pos], "%f", &rateAux);
				newUser->rating[ratePos] = rateAux;
				rateAux = -1.0f;
				break;
		}
		ratePos++;
    }
    
    pos++;
    
  }while(pos < size);
   
  return newUser;

}

int findAmmountOfRatesByUser(User *target, int nElements)
{
  int total = 0;

  for(int i=0; i<nElements; i++)
  {
    if(target->rating[i] != '?' && target->rating[i] != -1)
    {
      total++;
    }
  }

  return total;
}

// cálculo em GPU da média dos rates feitos pelo usuário
__global__ void cudaCalcUserAverageRate(float *ratings, int *dev_nElements, float result)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i < *dev_nElements)
	{
		result = 0.0f;
		int count = 0;
		
		for(int j=0; j < *dev_nElements; j++)
		{
			if(ratings[j] != '?' && ratings[j] != -1)
			{
				count++;
				result += ratings[j];
			}
		}
		
		result /= count;
	}
}


float calcUserAverageRate(User *target, int nElements)
{
  float average = 0.0f;
  int count = 0;

  for(int i=0; i<nElements; i++)
  {
    if(target->rating[i] != '?' && target->rating[i] != -1)
    {
      count++;
      average+= target->rating[i];
    }
  }

  average /= count;

  return average;
}

__global__ void cudaCalPearsonCorrelation(float *ratingsA, float *ratingsB, int *dev_nElements, float pearson)
{
	

	int i = threadIdx.x;
	

	if(i < *dev_nElements)
	{	
		int nRates = 0;
		float pearsonNum = 0.0f, pearsonDen = 0.0f; // pearson = 0.0f
		float tempA = 0.0f, tempB = 0.0f, tempC = 0.0f, tempD = 0.0f;
	
		if(ratingsA[i] == -1 || ratingsB[i] == -1)
		{
		  // DO NOTHING, someone hasn't evaluated an item
		}
		else // if both users have rated this item
		{
		  nRates++;
		  tempA += ratingsA[i]; //stores the Sum of the rates from A
		  tempB += ratingsB[i]; //stores the Sum of the rates from B
		  pearsonNum += ratingsA[i] * ratingsB[i];
		  tempC += powf(ratingsA[i], 2); // stores the Sum of the rate² from A
		  tempD += powf(ratingsB[i], 2); // stores the Sum of the rate² from B
		}

		pearsonNum -= (tempA * tempB) / nRates; // sumXY - ((sumX)(sumY)/n)
		tempC -= powf(tempA, 2) / nRates; // sumX² - ((sumX)²/n)
		tempD -= powf(tempB, 2) / nRates; // sumY² - ((sumY)²/n)
		pearsonDen = sqrtf(tempC * tempD);

		pearson = pearsonNum / pearsonDen;
	}
}

float calcPearsonCorrelation(User *a, User *b, int nElements)
{

  float pearson = 0.0f, pearsonNum = 0.0f, pearsonDen = 0.0f;
  float tempA = 0.0f, tempB = 0.0f, tempC = 0.0f, tempD = 0.0f;

  // int ratedItens[MAXITENS]; // Store the itens that are rated by both users | 1 = rated by both, 0 otherwise
  int nRates = 0;


  // loop that verify the itens that has been rated by each user and set the itens that can be used to calculate the similarity
  for(int i=0; i<nElements; i++)
  {
    if(a->rating[i] == -1 || b->rating[i] == -1)
    {
      continue; // someone hasn't evaluated an item
    }
    else // if both users have rated this item
    {
      // ratedItens[i] = 1;
      nRates++;
      tempA += a->rating[i]; //stores the Sum of the rates from A
      tempB += b->rating[i]; //stores the Sum of the rates from B
      pearsonNum += a->rating[i] * b->rating[i];
      tempC += pow(a->rating[i], 2); // stores the Sum of the rate² from A
      tempD += pow(b->rating[i], 2); // stores the Sum of the rate² from B
    }
  }

  pearsonNum -= (tempA * tempB) / nRates; // sumXY - ((sumX)(sumY)/n)
  tempC -= pow(tempA, 2) / nRates; // sumX² - ((sumX)²/n)
  tempD -= pow(tempB, 2) / nRates; // sumY² - ((sumY)²/n)
  pearsonDen = sqrt(tempC * tempD);

  pearson = pearsonNum / pearsonDen;
  //printf("Pearson entre %s e %s = %.2f\n", a->name, b-> name, pearson);
	
  return pearson;
}

float predictRateByUser(User **array, int a_size, User *desired, int itemID, int nElements)
{
	float pred = -0.5f, demRes = 0, numRes = 0, pearson = 0, des_average = 0, cur_average = 0, temp_average = 0;
	float *dev_ratings; 
	
	hipMalloc(&dev_ratings, nElements*sizeof(float));
	hipMemcpy(dev_ratings, desired->rating, nElements*sizeof(float), hipMemcpyHostToDevice);
	
	cudaCalcUserAverageRate<<<32, 1>>>(desired->rating, &nElements, des_average);
	
	hipFree(dev_ratings);
	
	for(int i=0; i<a_size; i++)
	{
		if(desired == array[i])
		{
			continue; // represents itself
		}
		else if(array[i]->rating[itemID] == -1)
		{			
			continue; // marks to skip in case it hasn't evaluate
		}
		else
		{
			cudaCalPearsonCorrelation<<<32, 1>>>(desired->rating, array[i]->rating, &nElements, pearson);
			if(pearson >= 0.7) 
			{
				demRes += pearson;
				hipMalloc(&dev_ratings, nElements*sizeof(float));
				hipMemcpy(dev_ratings, desired->rating, nElements*sizeof(float), hipMemcpyHostToDevice);
	
				cudaCalcUserAverageRate<<<32, 1>>>(desired->rating, &nElements, temp_average);

				cur_average = array[i]->rating[itemID] - temp_average;

				hipFree(dev_ratings);

				numRes += (pearson * cur_average);
			}
		}

	}
	
	pred = des_average + (numRes / demRes);
	
  return pred;
}


int main(int argc, char *argv[])
{

  int targetItemId = -1, nElements = 0, timesReaded = 0, maxLines = 0;
  char *targetUserName, readBuffer[1024], charBuffer, *filename;
  FILE *fp;
  User *targetUser = NULL;
  
  if(argc != 4)
  {
    fprintf(stderr,"Formato : %s Arquivo (string) Usuário (string) Item# (int)\n",argv[0]);
    return 1;
  }
  
  filename = argv[1];
  targetUserName = argv[2];
  sscanf(argv[3], "%d", &targetItemId);
  targetItemId--;
  
  fp = fopen(filename, "r");
  if(fp == NULL)
  {
    printf("File couldn't be loaded!\n");
    return 0;
  }
  
  while(feof(fp) == 0)
  {
    charBuffer = fgetc(fp);
    if(charBuffer == '\n')
    {
      maxLines++; // ammount of rows on the file
    }
  }

  rewind(fp); // reset buffer position
  maxLines--; // removes the header line

  fscanf(fp, "%s", readBuffer); // header line
  int size = strlen(readBuffer);

  for(int i=0; i<size; i++)
  {
    if(readBuffer[i] == ';')
    {
      nElements++; // stores the ammount of columns in the file
    }
  }
  
  if(targetItemId > nElements)
  {
	  fprintf(stderr,"Escopo incorreto!\nNão existe a coluna %d, este arquivo possui apenas %d colunas!\n\n", targetItemId, nElements );
	  return EXIT_FAILURE;
  }

  User *usersArray[maxLines];
  
  while(feof(fp) == 0 && timesReaded < maxLines)
  {
    fscanf(fp, "%s", readBuffer);

    if(readBuffer[0] != '\0')
    {
	  usersArray[timesReaded] = generateUser(readBuffer, nElements);
      timesReaded++;
      readBuffer[0] = '\0';
    }
  }
  
  fclose(fp);
  timesReaded = 0; // reseta o valor
  
  // finding the user
  for(int i=0; i<maxLines; i++)
  {
	  if(strcmp(usersArray[i]->name, targetUserName) == 0)
	  {
		  targetUser = usersArray[i];
	  }
	  // contabiliza o # de vezes que esse valor foi medido
	  if(usersArray[i]->rating[targetItemId] != -1)
	  {
		  timesReaded++;
	  }
  }
    
  if(targetUser == NULL)
  {
	  printf("Usuário não encontrado!\n");
	  return EXIT_FAILURE;
  }
  else
  {
	  printf("----------------------| Usuário %s |----------------------\n", targetUser->name);
	  printf("Avaliou %d Itens.\n", findAmmountOfRatesByUser(targetUser, nElements));
	  // usuário avaliou o item
	  if(targetUser->rating[targetItemId] != -1)
	  {
		  printf("Avaliou o Item solicitado: %.2f.\n", targetUser->rating[targetItemId]);
	  }
	  // usuário não avaliou o item
	  else
	  {
		  float pred1 = -1.0f, pred2 = -1.0f;
		  pred1 = predictRateByUser(usersArray, maxLines, targetUser, targetItemId, nElements);
		  printf("Não avaliou o Item Solicitado!!!\n\t|-Previsão por Usuário: %.2f\n\t|-Previsão por Item: %.2f\n", pred1, pred2);
	  }
	  printf("O Item Pesquisado foi avaliado %d vezes\n", timesReaded);
	  printf("--------------------------------------------------------\n");
	  
	  return EXIT_SUCCESS;
  }
   
}
